#include "hip/hip_runtime.h"
#include "Complex.cuh"
#include <math.h>

Complex Complex::W(int n) {
    Complex res(cos(2.0 * M_PI / n), sin(2.0 * M_PI / n));
    return res;
}