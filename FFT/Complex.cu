#include "hip/hip_runtime.h"
#include "Complex.cuh"
#include <math.h>

Complex W(int n) {
    Complex res(cos(2.0 * M_PI / n), sin(2.0 * M_PI / n));
    return res;
}