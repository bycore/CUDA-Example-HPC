#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""
#include "error.cuh"

void OddEvenSwitchSort(int* nums, int size);
int CheckFun(int* nums1, int* nums2, int n);
void SequentialBubbleSort(int* nums, int size);
__global__ void even_swapper(int* X, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i % 2 == 0 && i < N - 1) {
        if (X[i + 1] < X[i]) {
            int temp = X[i];
            X[i] = X[i + 1];
            X[i + 1] = temp;
        }
    }
}

__global__ void odd_swapper(int* X, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i % 2 != 0 && i < N - 1) {
        if (X[i + 1] < X[i]) {
            int temp = X[i];
            X[i] = X[i + 1];
            X[i + 1] = temp;
        }
    }
}
int main() {
    int size = 66535;
    int* nums1 = (int*)malloc(sizeof(int) * size + 1);
    int* nums2 = (int*)malloc(sizeof(int) * size + 1);
    srand(time(0));
    for (int i = 0; i < size; i++) {
        int num = rand();
        nums1[i] = num;
        nums2[i] = num;
    }
    printf("Number of numbers: %d\n", size);
    OddEvenSwitchSort(nums1, size);
    printf("GPU finish!\n");
    SequentialBubbleSort(nums2, size);
    printf("CPU finish!\n");
    int _ = CheckFun(nums1, nums2, size);
    if (_ == 0) {
        printf("successf!\n");
    } else {
        printf("wrong!\n");
    }
    free(nums1);
    free(nums2);
}

void OddEvenSwitchSort(int nums[], int size) {
    int* dNums;
    CHECK(hipMalloc((void**)&dNums, sizeof(int) * size));
    CHECK(hipMemcpy(dNums, nums, sizeof(int) * size, hipMemcpyHostToHost));

    dim3 threadPerBlock(32);
    dim3 blockNum((size + threadPerBlock.x - 1) / threadPerBlock.x);
    for (int i = 0; i < size; i++) {
        even_swapper<<<blockNum, threadPerBlock>>>(dNums, size);
        odd_swapper<<<blockNum, threadPerBlock>>>(dNums, size);
    }
    CHECK(hipMemcpy(nums, dNums, sizeof(int) * size, hipMemcpyDeviceToHost));

    CHECK(hipFree(dNums));
}

void SequentialBubbleSort(int* nums, int size) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size - 1; j++) {
            if (nums[j] > nums[j + 1]) {
                int temp = nums[j];
                nums[j] = nums[j + 1];
                nums[j + 1] = temp;
            }
        }
    }
}

int CheckFun(int* nums1, int* nums2, int n) {
    FILE* fp = NULL;
    fp = fopen("gz.txt", "w+");
    fprintf(fp, "%d\n", n);
    for (int i = 0; i < n; i++) {
        if (nums1[i] != nums2[i]) {
            //fprintf(fp, "%d:%d\t%d\n", i, nums1[i], nums2[i]);
            return 1;
        }
    }
    return 0;
}